#include <hip/hip_runtime.h>
#include <iostream>

int main() {
    int sharedMemPerBlock;
    hipDeviceGetAttribute(&sharedMemPerBlock, hipDeviceAttributeMaxSharedMemoryPerBlock, 0);
    std::cout << "Shared Memory Per Block: " << sharedMemPerBlock << " bytes" << std::endl;
    return 0;
}
